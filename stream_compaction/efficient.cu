#include "hip/hip_runtime.h"
#ifndef __HIPCC__
#define __HIPCC__
#endif // !__HIPCC__

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_cooperative_groups.h>
#include "common.h"
#include "efficient.h"

namespace cg = cooperative_groups;

namespace StreamCompaction {
	namespace Efficient {
		constexpr auto block_size = 512;
		using StreamCompaction::Common::PerformanceTimer;
		PerformanceTimer& timer()
		{
			static PerformanceTimer timer;
			return timer;
		}

		__global__
			void upsweep(int n, int* data, int d, int offset, int last_offset)
		{
			int idx = threadIdx.x + blockDim.x * blockIdx.x;
			if (idx >= n) return;
			if (idx % (2 << d) == 0)
			{
				data[idx + offset] += data[idx + last_offset];
			}
		}
		__global__
			void downsweep(int n, int* data, int d, int offset, int last_offset)
		{
			int idx = threadIdx.x + blockDim.x * blockIdx.x;
			if (idx >= n) return;
			int temp_down;
			if (idx % (2 << d) == 0)
			{
				temp_down = data[idx + last_offset];
				data[idx + last_offset] = data[idx + offset];
				data[idx + offset] += temp_down;
			}
		}

		void sweep_scan(int n, int* idata, int level, dim3 gridSize, dim3 blockSize)
		{
			int offset = 1;
			int last_offset = 0;
			int zero = 0;
			for (int d = 0; d < level; d++)
			{
				upsweep<<<gridSize, blockSize>>>(n, idata, d, offset, last_offset);
				last_offset = offset;
				offset += (2 << d);
				checkCUDAError("up sweep scan");

			}
			offset = last_offset;
			last_offset -= (1 << (level - 1));
			hipMemcpy(&idata[n-1], const_cast<const int*>(&zero), sizeof(int), hipMemcpyHostToDevice);
			checkCUDAError("memcpy");

			for (int d = level - 1; d >= 0; d--)
			{
				downsweep<<<gridSize, blockSize>>>(n, idata, d, offset, last_offset);
				offset = last_offset;
				last_offset -= (d == 0 ? 0 : (1 << (d - 1)));
				checkCUDAError("down sweep scan");

			}
		}

		__global__//DEPRECATED
			void sweep_scan(int n, int* odata, int* idata, int level)
		{
			//cg::grid_group cta = cg::this_grid();
			//printf("n=%d\n", cta.size());
			extern __shared__ int temp[];
			int idx = threadIdx.x + blockDim.x * blockIdx.x;
			if (idx >= n)return;
			int offset = 1;
			int last_offset = 0;
			int temp_down;
			temp[idx] = idata[idx];
			//UP-SWEEP
			//considering power-of-two first
			//1st round: 0,2,4,6 + 1 operates 1,3,5,7+0,2,4,6
			//2nd round: 0,4 + 3 operates 3,7+1,5
			//3rd round: 0 + 7 operates 7+3
			for (int d = 0; d < level; d++)
			{
				if (idx % (2 << d) == 0)
				{
					temp[idx + offset] += temp[idx + last_offset];
				}
				last_offset = offset;
				offset += (2 << d);
				__syncthreads();
				// you can't use threadfence to sync here
				//__threadfence()
				// cg::sync() needs LaunchCooperativeKernel
				//cta.sync();
			}
			//DOWN-SWEEP

			offset = last_offset;
			last_offset -= (1 << (level - 1));
			//now offset is 7, last_offset is 3, still consider power-of-two
			//
			if (idx == n - 1) temp[idx] = 0;
			for (int d = level - 1; d >= 0; d--)
			{
				if (idx % (2 << d) == 0)
				{
					temp_down = temp[idx + last_offset];
					temp[idx + last_offset] = temp[idx + offset];
					temp[idx + offset] += temp_down;
				}
				offset = last_offset;
				last_offset -= (d == 0 ? 0 : (1 << (d - 1)));
				__syncthreads();
				//__threadfence();
				//cta.sync();
			}
			odata[idx] = temp[idx];
		}

		/**
		 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
		 */
		void scan(int n, int* odata, const int* idata) {
			//size above 1024 need more blocks, but how to sync all the threads in a grid?
			//solution1 is to seperate the scan func, use for loop to do each op(implicit sync)
			//solution2 use cooperative groups. I give up, kernel launch always fails.
			int level = ilog2ceil(n);//incomplete binary tree
			//Here consider non-power-of-two, we need to pad 0s behind
			//So first round up n to nearest power-of-two
			int origin_n = n;
			n = 1 << level;

			dim3 blockSize(block_size);
			dim3 gridSize((n - 1) / blockSize.x + 1);

			int* g_odata;
			hipMalloc(&g_odata, n * sizeof(int));
			hipMemset(g_odata, 0, n * sizeof(int));
			//hipMemset(g_idata, 0, n * sizeof(int));
			//copy mem to device
			hipMemcpy(g_odata, idata, origin_n * sizeof(int), hipMemcpyHostToDevice);
			timer().startGpuTimer();
			// TODO
			//this O(n) algorithm contains 2 phase
			//Sweep
			//sweep_scan << <gridSize, blockSize, n * sizeof(int), 0 >> > (n, g_odata, g_odata, level);
			sweep_scan(n, g_odata, level, gridSize, blockSize);
			checkCUDAError("sweep scan");
			//void* kernelArgs[] = { &n, g_odata, g_idata, &level };
			//hipLaunchCooperativeKernel((void *)sweep_scan, gridSize, blockSize, kernelArgs, n * sizeof(int), 0);
			//checkCUDAError("Error launching cooperative kernel");
			timer().endGpuTimer();
			//copy mem to host
			hipMemcpy(odata, g_odata, origin_n * sizeof(int), hipMemcpyDeviceToHost);
			hipFree(g_odata);
		}

		/**
		 * Performs stream compaction on idata, storing the result into odata.
		 * All zeroes are discarded.
		 *
		 * @param n      The number of elements in idata.
		 * @param odata  The array into which to store elements.
		 * @param idata  The array of elements to compact.
		 * @returns      The number of elements remaining after compaction.
		 */
		int compact(int n, int* odata, const int* idata) {
			int level = ilog2ceil(n);
			int origin_n = n;
			int compacted_num;
			n = 1 << level;
			dim3 blockSize(block_size);
			dim3 gridSize((n - 1) / blockSize.x + 1);
			int* g_odata, * g_idata, * g_bools, *g_indice;
			hipMalloc(&g_odata, n * sizeof(int));
			hipMalloc(&g_idata, n * sizeof(int));
			hipMalloc(&g_bools, n * sizeof(int));
			hipMalloc(&g_indice, n * sizeof(int));
			hipMemset(g_idata, 0, n * sizeof(int));
			hipMemset(g_bools, 0, n * sizeof(int));
			hipMemset(g_odata, 0, n * sizeof(int));
			hipMemset(g_indice, 0, n * sizeof(int));
			//copy mem to device
			hipMemcpy(g_idata, idata, origin_n * sizeof(int), hipMemcpyHostToDevice);

			timer().startGpuTimer();
			// TODO
			StreamCompaction::Common::kernMapToBoolean << <gridSize, blockSize >> > (n, g_bools, g_idata);
			//sweep_scan << <gridSize, blockSize, n * sizeof(int), 0 >> > (n, g_odata, g_bools, level);
			hipMemcpy(g_indice, g_bools, origin_n * sizeof(int), hipMemcpyDeviceToDevice);
			sweep_scan(n, g_indice, level, gridSize, blockSize);
			StreamCompaction::Common::kernScatter << <gridSize, blockSize >> > (n, g_odata, g_idata, g_bools, g_indice);
			timer().endGpuTimer();
			//hipDeviceSynchronize();

			if (origin_n == n) hipMemcpy(&compacted_num, &g_indice[origin_n - 1], sizeof(int), hipMemcpyDeviceToHost);
			else hipMemcpy(&compacted_num, &g_indice[origin_n], sizeof(int), hipMemcpyDeviceToHost);
			hipMemcpy(odata, g_odata, n * sizeof(int), hipMemcpyDeviceToHost);

			hipFree(g_odata);
			hipFree(g_idata);
			hipFree(g_bools);
			hipFree(g_indice);

			return compacted_num;
		}
	}
}
