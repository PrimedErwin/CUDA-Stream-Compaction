#include "hip/hip_runtime.h"
#ifndef __HIPCC__
#define __HIPCC__
#endif // !__HIPCC__

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/scan.h>
#include "hac.h"

namespace cg = cooperative_groups;


namespace StreamCompaction {
	namespace HAC {
		constexpr int block_size = 512;

		using StreamCompaction::Common::PerformanceTimer;
		PerformanceTimer& timer()
		{
			static PerformanceTimer timer;
			return timer;
		}

		__device__
			int group_need_work(int* work_group_index, int size, int groupID, int work_group_size)
		{
			int minval = work_group_index[size - 1];
			for (int i = size-1; i >= 0; i--)
			{
				minval = (minval < work_group_index[i] && work_group_index[i] - minval < work_group_size) ? minval : work_group_index[i];
				if (groupID == work_group_index[i])
				{
					return minval;
				}
			}
			return 0;
		}

		__global__
			void tiled_scan(int n, int* odata, int* idata, int level)
		{
			extern __shared__ int work_group_index[];
			int idx = threadIdx.x + blockDim.x * blockIdx.x;
			if (idx >= n) return;
			cg::thread_block cta = cg::this_thread_block();
			auto tile32 = cg::tiled_partition<32>(cta);
			int group_rank = tile32.meta_group_rank();


			//odata[idx] = cg::exclusive_scan(tile32, idata[idx]);
			odata[idx] = cg::exclusive_scan(tile32, idata[idx]);
			cta.sync();

			//okay let's upsweep
			//we need half of the threads each time, assume here are 8 groups
			//1st round: 1,3,5,7 operates 0,2,4,6
			//2nd round: 2,3,6,7 operates 0,1,4,5
			//3rd round: 4,5,6,7 operates 0,1,2,3
			for (int d = 0; d < level; d++)
			{
				//if (group_rank % (2<<d) == 1)
				//{
				//	idata[idx] += idata[group_rank * 32 - 1];
				//}

				//1st round it's 1 group, 2nd is 2 groups work together
				int work_group_size = (1 << d);
				//int work_group_num = block_size / 32 / 2;
				int temp_index = 0;
				int work_index = 0;
				for (int i = block_size / 32 - 1; i >= 0; i -= work_group_size)
				{
					for (int j = work_group_size; j > 0; j--)
					{
						if (threadIdx.x == 0)work_group_index[temp_index++] = i--;
						else temp_index++;
						//if (idx == 0)printf("%d round, %d index, %d\n", d, temp_index - 1, i + 1);
					}
				}
				cta.sync();
				//now all the groups that need to work has been stored in array
				if (work_index = group_need_work(work_group_index, temp_index, group_rank, work_group_size))
				{
					//if(tile32.thread_rank()==0) printf("%d Group %d in %d\n", d, group_rank, work_index);
					//minus work_gourp_size to get the group rank that needs operate
					odata[idx] += odata[work_index * 32 - 1]
						+idata[work_index * 32 - 1];
				}
				cta.sync();
			}
		}

		//calculate work_group_index for grid-level recursive sort
		__global__
			void grid_work_group_index(int* work_group_index, int d, int* temp_index, dim3 gridSize)
		{
			temp_index[0] = 0;
			int work_group_size = (1 << d);
			for (int i = gridSize.x - 1; i >= 0; i -= work_group_size)
			{
				for (int j = work_group_size; j > 0; j--)
				{
					work_group_index[temp_index[0]++] = i--;
					//printf("%d round, %d index, %d\n", d, temp_index[0] - 1, i + 1);
				}
			}

		}

		//we have a lot of blocks with each one full of 512 prefix sums
		__global__
			void block_scan(int n, int* odata, int* idata, int* work_group_index, int* temp_index, int d)
		{
			int idx = threadIdx.x + blockDim.x * blockIdx.x;
			if (idx >= n) return;
			int group_rank = idx / block_size;
			int work_index = 0;
			int work_group_size = (1 << d);
			if (work_index = group_need_work(work_group_index, temp_index[0], group_rank, work_group_size))
			{
				//if(tile32.thread_rank()==0) printf("%d Group %d in %d\n", d, group_rank, work_index);
				//minus work_gourp_size to get the group rank that needs operate
				odata[idx] += odata[work_index * block_size - 1]
					+ idata[work_index * block_size - 1];
			}

		}






		void scan(int n, int* odata, const int* idata)
		{
			int level = ilog2ceil(n);//incomplete binary tree
			int origin_n = n;
			n = 1 << level;
			int level_32 = ilog2ceil(block_size / 32);

			dim3 blockSize(block_size);
			dim3 gridSize((n - 1) / blockSize.x + 1);

			int level_block_512 = ilog2ceil(gridSize.x);
			int* g_odata, * g_idata, *g_work_group_grid, *g_temp_index;
			hipMalloc(&g_odata, n * sizeof(int));
			hipMalloc(&g_work_group_grid, level_block_512/2 * sizeof(int));
			hipMalloc(&g_idata, n * sizeof(int));
			hipMalloc(&g_temp_index,  sizeof(int));
			hipMemset(g_odata, 0, n * sizeof(int));
			hipMemset(g_idata, 0, n * sizeof(int));

			hipMemcpy(g_idata, idata, origin_n * sizeof(int), hipMemcpyHostToDevice);

			timer().startGpuTimer();
			tiled_scan << <gridSize, blockSize, (block_size / 64) * sizeof(int), 0 >> > (n, g_odata, g_idata, level_32);
			checkCUDAError("tiled_scan");
			//above we get prefix sums by block
			//the following performs a block scan
			for (int d = 0; d < level_block_512; d++)
			{
				grid_work_group_index<<<1,1>>>(g_work_group_grid, d, g_temp_index, gridSize);
				block_scan<<<gridSize, blockSize>>>(n, g_odata, g_idata, g_work_group_grid, g_temp_index, d);
				checkCUDAError("block scan");
			}
			timer().endGpuTimer();

			hipMemcpy(odata, g_odata, origin_n * sizeof(int), hipMemcpyDeviceToHost);
			checkCUDAError("hac_memcpy");
			hipFree(g_idata);
			hipFree(g_odata);
			hipFree(g_work_group_grid);
			hipFree(g_temp_index);

		}

		int compact(int n, int* odata, const int* idata)
		{

			return -1;
		}


	}
}