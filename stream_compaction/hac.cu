#include "hip/hip_runtime.h"
#ifndef __HIPCC__
#define __HIPCC__
#endif // !__HIPCC__

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/scan.h>
#include "hac.h"

namespace cg = cooperative_groups;


namespace StreamCompaction {
	namespace HAC {
		constexpr int block_size = 512;

		using StreamCompaction::Common::PerformanceTimer;
		PerformanceTimer& timer()
		{
			static PerformanceTimer timer;
			return timer;
		}

		__device__
			int group_need_work(int* work_group_index, int size, int groupID)
		{
			for (int i = 0; i < size; i++)
			{
				if (groupID == work_group_index[i])
				{
					return 1;
				}
			}
			return 0;
		}

		__global__
			void tiled_scan(int n, int* odata, int* idata, int level)
		{
			extern __shared__ int work_group_index[];
			int idx = threadIdx.x + blockDim.x * blockIdx.x;
			if (idx >= n) return;
			cg::thread_block cta = cg::this_thread_block();
			auto tile32 = cg::tiled_partition<32>(cta);
			int group_rank = tile32.meta_group_rank();

			if (blockIdx.x == 0)
			{
				if (group_rank == 0)
				{
					//odata[idx] = cg::exclusive_scan(tile32, idata[idx]);
					idata[idx] = cg::exclusive_scan(tile32, idata[idx]);
				}
				else
				{
					//odata[idx] = cg::inclusive_scan(tile32, idata[idx]);
					idata[idx] = cg::inclusive_scan(tile32, idata[idx]);

				}
				cta.sync();
			}
			else
			{
				idata[idx] = cg::inclusive_scan(tile32, idata[idx]);

			}

			//okay let's upsweep
			//we need half of the threads each time, assume here are 8 groups
			//1st round: 1,3,5,7 operates 0,2,4,6
			//2nd round: 2,3,6,7 operates 0,1,4,5
			//3rd round: 4,5,6,7 operates 0,1,2,3
			for (int d = 0; d < level; d++)
			{
				//if (group_rank % (2<<d) == 1)
				//{
				//	idata[idx] += idata[group_rank * 32 - 1];
				//}

				//1st round it's 1 group, 2nd is 2 groups work together
				int work_group_size = (1 << d);
				//int work_group_num = block_size / 32 / 2;
				int temp_index = 0;
				for (int i = block_size / 32 - 1; i >= 0; i -= work_group_size)
				{
					for (int j = work_group_size; j > 0; j--)
					{
						work_group_index[temp_index++] = i--;
						//if (idx == 0)printf("%d round, %d index, %d\n", d, temp_index - 1, i + 1);
					}
				}
				//now all the groups that need to work has been stored in array
				if (group_need_work(work_group_index, temp_index, group_rank))
				{
					//if(tile32.thread_rank()==0) printf("%d Group %d in \n", d, group_rank);
					//minus work_gourp_size to get the group rank that needs operate
					idata[idx] += idata[(group_rank-work_group_size+1) * 32 - 1];
				}
				cta.sync();
			}
		}

		//we have a lot of blocks with each one full of prefix sums
		__global__
			void block_scan()
		{

		}






		void scan(int n, int* odata, const int* idata)
		{
			int level = ilog2ceil(n);//incomplete binary tree
			int origin_n = n;
			n = 1 << level;
			int level_32 = ilog2ceil(block_size / 32);

			dim3 blockSize(block_size);
			dim3 gridSize((n - 1) / blockSize.x + 1);

			int level_block_512 = ilog2ceil(gridSize.x);
			int* g_odata, * g_idata;
			hipMalloc(&g_odata, n * sizeof(int));
			hipMalloc(&g_idata, n * sizeof(int));
			hipMemset(g_odata, 0, n * sizeof(int));
			hipMemset(g_idata, 0, n * sizeof(int));

			hipMemcpy(g_idata, idata, origin_n * sizeof(int), hipMemcpyHostToDevice);

			timer().startCpuTimer();
			tiled_scan << <gridSize, blockSize, (block_size / 64) * sizeof(int), 0 >> > (n, g_odata, g_idata, level_32);
			checkCUDAError("tiled_scan");
			//above we get prefix sums by block
			//the following performs a block scan

			timer().endCpuTimer();

			hipMemcpy(odata, g_idata, origin_n * sizeof(int), hipMemcpyDeviceToHost);
			checkCUDAError("hac_memcpy");
			hipFree(g_idata);
			hipFree(g_odata);

		}

		int compact(int n, int* odata, const int* idata)
		{

			return -1;
		}


	}
}